
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE


__global__ void histo_kernel(unsigned int* input, unsigned int *bins, unsigned int num_elements,
        unsigned int num_bins)
{
	__shared__ unsigned int histo_private[4096];
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < 4096)
		histo_private[i] = 0;
	__syncthreads();
	
	//int stride = blockDim.x * gridDim.x;
        while(i < num_elements){
                atomicAdd(&histo_private[input[i]], 1);
		//i += stride;
	}
	__syncthreads();
	
	if(i < 4096)
		atomicAdd(&bins[i], histo_private[i]);
        
}









/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // INSERT CODE HERE
        
        // Initialize thread block and kernel grid dimensions ---------------------
        const unsigned int BLOCK_SIZE = 128;

	dim3 dim_grid = (num_elements + BLOCK_SIZE - 1)/BLOCK_SIZE;
	dim3 dim_block = BLOCK_SIZE;
	
        // Invoke CUDA kernel -----------------------------------------------------
	histo_kernel<<<dim_grid, dim_block>>>(input, bins, num_elements, num_bins);


}


