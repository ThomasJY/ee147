#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE


__global__ void histo_kernel(unsigned int* input, unsigned int *bins, unsigned int num_elements,
        unsigned int num_bins)
{
	__shared__ unsigned int histo_private[7];
	
	if(threadIdx.x < 4096)
		histo_private[threadidx.x] = 0;
	__syncthreads();
	
        int i = blockIdx.x*blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
        while(i < num_elements){
                atomicAdd(&histo_private[input[i]], 1);
		i += stride;
	}
	__syncthreads();
	
	if(threadIdx.x < 4096)
		atomicAdd(&bins[threadIdx.x], histo_private[threadIdx.x]);
        
}









/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // INSERT CODE HERE
        
        // Initialize thread block and kernel grid dimensions ---------------------
        const unsigned int BLOCK_SIZE = 128;

	dim3 dim_grid = (num_elements + BLOCK_SIZE - 1)/BLOCK_SIZE;
	dim3 dim_block = BLOCK_SIZE;
	
        // Invoke CUDA kernel -----------------------------------------------------
	histo_kernel<<<dim_grid, dim_block>>>(input, bins, num_elements, num_bins);


}


